#include "hip/hip_runtime.h"

#include <stdio.h>
#include "debug.h"
#include "kernels.cuh"
#include <hip/hip_math_constants.h>

__device__ const int blockSize = 256;
__device__ const int warp = 32;
__device__ const int stackSize = 64;
__device__ const float eps2 = 0.025;
__device__ const float theta = 0.5;


__global__ void set_draw_array_kernel(float *ptr, float *x, float *y, int n)
{
	int index = threadIdx.x + blockDim.x*blockIdx.x;
	
	if(index < n){
		ptr[2*index] = x[index];
		ptr[2*index+1] = y[index];

	}
}


__global__ void reset_arrays_kernel(int *mutex, float *x, float *y, float *mass, int *count, int *start, int *sorted, int *child, int *index, float *left, float *right, float *bottom, float *top, int n, int m)
{
	int bodyIndex = threadIdx.x + blockDim.x*blockIdx.x;
	int stride = blockDim.x*gridDim.x;
	int offset = 0;

	// reset quadtree arrays
	while(bodyIndex + offset < m){  
#pragma unroll 4
		for(int i=0;i<4;i++){
			child[(bodyIndex + offset)*4 + i] = -1;
		}
		if(bodyIndex + offset < n){
			count[bodyIndex + offset] = 1;
		}
		else{
			x[bodyIndex + offset] = 0;
			y[bodyIndex + offset] = 0;
			mass[bodyIndex + offset] = 0;
			count[bodyIndex + offset] = 0;
		}
		start[bodyIndex + offset] = -1;
		sorted[bodyIndex + offset] = 0;
		offset += stride;
	}

	if(bodyIndex == 0){
		*mutex = 0;
		*index = n;
		*left = HIP_INF_F;
		*right = -HIP_INF_F;
		*bottom = HIP_INF_F;
		*top = -HIP_INF_F;
	}
}

  
__global__ void compute_bounding_box_kernel(int *mutex, float *x, float *y, volatile float *left, volatile float *right, volatile float *bottom, volatile float *top, int n)
{
	int index = threadIdx.x + blockDim.x*blockIdx.x;
	int stride = blockDim.x*gridDim.x;
	float x_min = x[index];
	float x_max = x[index];
	float y_min = y[index];
	float y_max = y[index];
	
	__shared__ float left_cache[blockSize];
	__shared__ float right_cache[blockSize];
	__shared__ float bottom_cache[blockSize];
	__shared__ float top_cache[blockSize];


	int offset = stride;
	while(index + offset < n){
		x_min = fminf(x_min, x[index + offset]);
		x_max = fmaxf(x_max, x[index + offset]);
		y_min = fminf(y_min, y[index + offset]);
		y_max = fmaxf(y_max, y[index + offset]);
		offset += stride;
	}

	left_cache[threadIdx.x] = x_min;
	right_cache[threadIdx.x] = x_max;
	bottom_cache[threadIdx.x] = y_min;
	top_cache[threadIdx.x] = y_max;

	__syncthreads();

	//////////////////////////
	// BLOCK-WISE REDUCTION //
	//////////////////////////

	// NOTE: This could be done by warps

	// assumes blockDim.x is a power of 2!
	int i = blockDim.x/2;
	while(i != 0){
		if(threadIdx.x < i){
			left_cache[threadIdx.x]   = fminf(left_cache[threadIdx.x], left_cache[threadIdx.x + i]);
			right_cache[threadIdx.x]  = fmaxf(right_cache[threadIdx.x], right_cache[threadIdx.x + i]);
			bottom_cache[threadIdx.x] = fminf(bottom_cache[threadIdx.x], bottom_cache[threadIdx.x + i]);
			top_cache[threadIdx.x]    = fmaxf(top_cache[threadIdx.x], top_cache[threadIdx.x + i]);
		}
		__syncthreads();
		i /= 2;
	}

	/////////////////////
	// FINAL REDUCTION //
	/////////////////////

	//NOTE: threadIdx.x == 0 in each block performs final reduction using atomics

	// How the lock works
	// -If a thread has the lock, the mutex will be 1, and the thread loops (spin lock)
	// -If a thread does not have the lock, it takes the lock and is done

	if(threadIdx.x == 0){
		while (atomicCAS(mutex, 0 ,1) != 0); // lock
		*left = fminf(*left, left_cache[0]);
		*right = fmaxf(*right, right_cache[0]);
		*bottom = fminf(*bottom, bottom_cache[0]);
		*top = fmaxf(*top, top_cache[0]);
		atomicExch(mutex, 0); // unlock
	}
}


__global__ void build_tree_kernel(volatile float *x, volatile float *y, volatile float *mass, volatile int *count,
									int *start, volatile int *child, int *index,
									const float *left, const float *right, const float *bottom, const float *top,
									const int n, const int m)
{
	/*
	index:	a global index start at n
	n:		the number of bodies
	m:		the number of possible nodes
	*/

	int bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = blockDim.x*gridDim.x;
	int offset = 0;
	bool newBody = true;

	// build quadtree
	float l; 
	float r; 
	float b; 
	float t;
	int childPath;
	int temp;
	offset = 0;
	while((bodyIndex + offset) < n){

		if(newBody){
			newBody = false;
			//Top/Down Traversal: All particles start in one of the top 4 quads

			l = *left;
			r = *right;
			b = *bottom;
			t = *top;

			//Check body location within the top 4 nodes
			temp = 0;
			childPath = 0;
			if(x[bodyIndex + offset] < 0.5*(l+r)){
				childPath += 1;
				r = 0.5*(l+r);
			}
			else{
				l = 0.5*(l+r);
			}
			if(y[bodyIndex + offset] < 0.5*(b+t)){
				childPath += 2;
				t = 0.5*(t+b);
			}
			else{
				b = 0.5*(t+b);
			}
		}

		//Set childIndex, which could be after mutliple loops
		int childIndex = child[temp*4 + childPath];

		// traverse tree until we hit leaf node (could be allocated or not)

		//NOTE: childIndex >= n means we are in a cell not a leaf
		// You could also land in an unallocated (-1) or locked (-2) node
		while(childIndex >= n){
			//Check body location within the 4 quads of this node
			temp = childIndex;
			childPath = 0;
			if(x[bodyIndex + offset] < 0.5*(l+r)){
				childPath += 1;
				r = 0.5*(l+r);
			}
			else{
				l = 0.5*(l+r);
			}
			if(y[bodyIndex + offset] < 0.5*(b+t)){
				childPath += 2;
				t = 0.5*(t+b);
			}
			else{
				b = 0.5*(t+b);
			}

			//Update the Centroid in this cell
			atomicAdd((float*)&x[temp], mass[bodyIndex + offset]*x[bodyIndex + offset]);
			atomicAdd((float*)&y[temp], mass[bodyIndex + offset]*y[bodyIndex + offset]);
			//Increment total mass in this cell
			atomicAdd((float*)&mass[temp], mass[bodyIndex + offset]);
			//Increment body count within this cell
			atomicAdd((int*)&count[temp], 1);

			//Advance to child of this cell
			childIndex = child[4*temp + childPath];
		}

		// Check if child is already locked i.e. childIndex == -2
		if(childIndex != -2){
			//Acquire lock
			int locked = temp*4 + childPath;
			if(atomicCAS((int*)&child[locked], childIndex, -2) == childIndex){
				//If unallocated, insert body and unlock
				if(childIndex == -1){
					//The initial assignment of childIndex -1 -> body Idx
					child[locked] = bodyIndex + offset;
				}
				else{

					//Sets max on number of internal nodes
					int patch = 4*n;
					while(childIndex >= 0 && childIndex < n){

						//NOTE: the childIndex < n should never obtain.
						// childIndex should always be -1, unallocated, or >=0, allocated

						//Create a new cell, starting at index n
						int cell = atomicAdd(index,1);
						patch = min(patch, cell);	// ??? this will be patch == cell until cell >= 4*n

						//Re-assign child from body Index to new cell index
						if(patch != cell){
							child[4*temp + childPath] = cell;
						}

						// insert old particle into new cell
						childPath = 0;
						if(x[childIndex] < 0.5*(l+r)){
							childPath += 1;
						}
						if(y[childIndex] < 0.5*(b+t)){
							childPath += 2;
						}

						if(DEBUG){
							// if(cell >= 2*n){
							if(cell >= m){
								printf("%s\n", "error cell index is too large!!");
								printf("cell: %d\n", cell);
							}
						}

						//Update the Centroid in this new cell with old particle
						x[cell] += mass[childIndex]*x[childIndex];
						y[cell] += mass[childIndex]*y[childIndex];
						//Increment total mass in this new cell with old particle
						mass[cell] += mass[childIndex];
						//Increments body count within this cell with old particle
						count[cell] += count[childIndex];
						//Re-assign old particle to subtree entry
						child[4*cell + childPath] = childIndex;

						start[cell] = -1;

						// insert new particle
						temp = cell;
						childPath = 0;
						if(x[bodyIndex + offset] < 0.5*(l+r)){
							childPath += 1;
							r = 0.5*(l+r);
						}
						else{
							l = 0.5*(l+r);
						}
						if(y[bodyIndex + offset] < 0.5*(b+t)){
							childPath += 2;
							t = 0.5*(t+b);
						}
						else{
							b = 0.5*(t+b);
						}
						//Update the Centroid in this new cell with new particle
						x[cell] += mass[bodyIndex + offset]*x[bodyIndex + offset];
						y[cell] += mass[bodyIndex + offset]*y[bodyIndex + offset];
						//Increment total mass in this new cell with new particle
						mass[cell] += mass[bodyIndex + offset];
						//Increments body count within this cell with new particle
						count[cell] += count[bodyIndex + offset];

						//Set to value of child at this entry, which could be:
						// -1 == break
						// a body index, meaning the need to further subdivide
						childIndex = child[4*temp + childPath]; 
					}

					//This means childIndex is set to -1, unallocated, so allocated as body Index
					child[4*temp + childPath] = bodyIndex + offset;

					__threadfence();  // Ensures all writes to global memory are complete before lock is released

					//Now this locked Index is a cell
					child[locked] = patch;
				}	// if(childIndex == -1): first assignment to body or not

				offset += stride;
				newBody = true;
			}	//if(atomicCAS((int*)&child[locked], childIndex, -2) == childIndex)

		}	//if(childIndex != -2): locked already or not. If locked, go around again

		// Wait for threads in block to release locks to reduce memory pressure
		__syncthreads(); // not strictly needed for correctness
	}
}



__global__ void centre_of_mass_kernel(float *x, float *y, float *mass, int *index, int n)
{
	int bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = blockDim.x*gridDim.x;
	int offset = 0;

	bodyIndex += n;
	while(bodyIndex + offset < *index){
		x[bodyIndex + offset] /= mass[bodyIndex + offset];
		y[bodyIndex + offset] /= mass[bodyIndex + offset];

		offset += stride;
	}
}



__global__ void sort_kernel(int *count, int *start, int *sorted, int *child, int *index, int n)
{
	int bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = blockDim.x*gridDim.x;
	int offset = 0;

	int s = 0;
	if(threadIdx.x == 0){
		for(int i=0;i<4;i++){
			int node = child[i];

			if(node >= n){  // not a leaf node
				start[node] = s;
				s += count[node];
			}
			else if(node >= 0){  // leaf node
				sorted[s] = node;
				s++;
			}
		}
	}

	int cell = n + bodyIndex;
	int ind = *index;
	while((cell + offset) < ind){
		s = start[cell + offset];
	
		if(s >= 0){

			for(int i=0;i<4;i++){
				int node = child[4*(cell+offset) + i];

				if(node >= n){  // not a leaf node
					start[node] = s;
					s += count[node];
				}
				else if(node >= 0){  // leaf node
					sorted[s] = node;
					s++;
				}
			}
			offset += stride;
		}
	}
}



__global__ void compute_forces_kernel(float* x, float *y, float *vx, float *vy, float *ax, float *ay, float *mass, int *sorted, int *child, float *left, float *right, int n, float g)
{
	int bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = blockDim.x*gridDim.x;
	int offset = 0;

	__shared__ float depth[stackSize*blockSize/warp]; 
	__shared__ int stack[stackSize*blockSize/warp];  // stack controled by one thread per warp 

	float radius = 0.5*(*right - (*left));

	// need this in case some of the first four entries of child are -1 (otherwise jj = 3)
	int jj = -1;                 
	for(int i=0;i<4;i++){       
		if(child[i] != -1){     
			jj++;               
		}                       
	}

	int counter = threadIdx.x % warp;
	int stackStartIndex = stackSize*(threadIdx.x / warp);
	while(bodyIndex + offset < n){
		int sortedIndex = sorted[bodyIndex + offset];

		float pos_x = x[sortedIndex];
		float pos_y = y[sortedIndex];
		float acc_x = 0;
		float acc_y = 0; 

		// initialize stack
		int top = jj + stackStartIndex;
		if(counter == 0){
			int temp = 0;
			for(int i=0;i<4;i++){
				if(child[i] != -1){
					stack[stackStartIndex + temp] = child[i];
					depth[stackStartIndex + temp] = radius*radius/theta;
					temp++;
				}
				// if(child[i] == -1){
				// 	printf("%s %d %d %d %d %s %d\n", "THROW ERROR!!!!", child[0], child[1], child[2], child[3], "top: ",top);
				// }
				// else{
				// 	stack[stackStartIndex + temp] = child[i];
				// 	depth[stackStartIndex + temp] = radius*radius/theta;
				// 	temp++;	
				// }
			}
		}

		__syncthreads();

		// while stack is not empty
		while(top >= stackStartIndex){
			int node = stack[top];
			float dp = 0.25*depth[top];
			// float dp = depth[top];
			for(int i=0;i<4;i++){
				int ch = child[4*node + i];

				//__threadfence();
			
				if(ch >= 0){
					float dx = x[ch] - pos_x;
					float dy = y[ch] - pos_y;
					float r = dx*dx + dy*dy + eps2;
					if(ch < n /*is leaf node*/ || __all(dp <= r)/*meets criterion*/){
						r = rsqrt(r);
						float f = mass[ch] * r * r * r;

						acc_x += f*dx;
						acc_y += f*dy;
					}
					else{
						if(counter == 0){
							stack[top] = ch;
							depth[top] = dp;
							// depth[top] = 0.25*dp;
						}
						top++;
						//__threadfence();
					}
				}
			}

			top--;
		}

		ax[sortedIndex] = acc_x;
		ay[sortedIndex] = acc_y;

		offset += stride;

		__syncthreads();
	}
}



__global__ void update_kernel(float *x, float *y, float *vx, float *vy, float *ax, float *ay, int n, float dt, float d){
	int bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = blockDim.x*gridDim.x;
	int offset = 0;

	while(bodyIndex + offset < n){
		vx[bodyIndex + offset] += dt*ax[bodyIndex + offset]; 
		vy[bodyIndex + offset] += dt*ay[bodyIndex + offset]; 

		x[bodyIndex + offset] += d*dt*vx[bodyIndex + offset]; 
		y[bodyIndex + offset] += d*dt*vy[bodyIndex + offset]; 

		offset += stride;
	} 
}



__global__ void copy_kernel(float *x, float *y, float *out, int n)
{
	int bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = blockDim.x*gridDim.x;
	int offset = 0;

	while(bodyIndex + offset < n){
		out[2*(bodyIndex + offset)] = x[bodyIndex + offset];
		out[2*(bodyIndex + offset) + 1] = y[bodyIndex + offset];

		offset += stride;
	}
}
